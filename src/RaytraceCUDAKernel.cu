#include "hip/hip_runtime.h"
#include "gpu.cuh"
#include <iostream>

__global__ void raytrace_k(float *hmap, float *poses_inds, float *max_pts_inds,
                  		   bool *mask, int N, int W, int H, int P) {

	// Adapted from http://playtechs.blogspot.com/2007/03/raytracing-on-grid.html

	// Get indices
	int i = blockIdx.x * blockDim.x + threadIdx.x; // Ray index (one thread per ray)
	int j = int(floor(i / N)); // Pose index
	if (i > (N * P)) return;

	/***** raytrace through ray (this is sloppy repetitive code but I'm lazy) *****/

	// Pull ending values for specific ray
	int x_max = (int)max_pts_inds[3*i];
	int y_max = (int)max_pts_inds[3*i+1];
	float z_max = max_pts_inds[3*i+2];

	// Pull pose values
	int pose_x = poses_inds[3*j];
	int pose_y = poses_inds[3*j+1];
	float pose_z = poses_inds[3*j+2];
	int z_start = int(floor(pose_z));
	
	// Setup
	int dx = abs(x_max - pose_x);
	int dy = abs(y_max - pose_y);
	float dz = fabs(z_max - pose_z);

	double dt_dx = 1.0 / dx;
	double dt_dy = 1.0 / dy;
	double dt_dz = 1.0 / dz;

	int n = 1;
	int x_inc, y_inc, z_inc;
	double t_next_x, t_next_y, t_next_z;

	// define initial variables based on cases
	// x
	if (dx == 0){
		x_inc = 0;
		t_next_x = 1000.0;
	} else if (x_max > pose_x) {
		x_inc = 1;
		n += x_max - pose_x;
		t_next_x = dt_dx;
	} else {
		x_inc = -1;
		n += pose_x - x_max;
		t_next_x = -dt_dx;
	}

	// y
	if (dy == 0){
		y_inc = 0;
		t_next_y = 1000.0;
	} else if (y_max > pose_y) {
		y_inc = 1;
		n += y_max - pose_y;
		t_next_y = dt_dy;
	} else {
		y_inc = -1;
		n += pose_y - y_max;
		t_next_y = -dt_dy;
	}

	// z
	if (dz == 0){
		z_inc = 0;
		t_next_z = 1000.0;
	} else if (z_max > pose_z) {
		z_inc = 1;
		n += int(floor(z_max)) - pose_z;
		t_next_z = (z_start + 1 - pose_z) * dt_dz;
	} else {
		z_inc = -1;
		n += pose_z - int(floor(z_max));
		t_next_z = (pose_z - z_start) * dt_dz;
	}

	// loop through ray and update mask as necessary
	float z_curr = z_start;
	int x_grid = pose_x;
	int y_grid = pose_y;
	int z_grid = z_start;
	float hmap_z;
	double t = 0;

	for (; n > 0; --n){

		// check if current grid index is valid (return if not)
		if (x_grid >= W || x_grid < 0 || y_grid >= H || y_grid < 0) return;

		// Get current x, y, and z given t
		z_curr = z_start + t * z_inc * dz;
		
		// check if current position is above ground (update mask and return if not)
		hmap_z = hmap[y_grid + x_grid * H];
		if (hmap_z >= z_curr) {
			mask[y_grid + x_grid * H] = false;
			return;
		}

		// take a step along the ray
		if (t_next_x < t_next_y && t_next_x < t_next_z) {

			// x is min
			x_grid += x_inc;
			t = t_next_x;
			t_next_x += dt_dx;

		} else if(t_next_y < t_next_x && t_next_y < t_next_z) {

			// y is min
			y_grid += y_inc;
			t = t_next_y;
			t_next_y += dt_dy;

		} else {

			// z is min
			z_grid += z_inc;
			t = t_next_z;
			t_next_z += dt_dz;

		}
	}
}

void RaytraceCUDAKernel(float *hmap, float *poses_inds, float *max_pts_inds,
						bool *mask, int N, int W, int H, int P, hipStream_t stream) {
						
	// create shared arrays for heightmap and mask
	float *d_hmap, *d_pose, *d_max_pts;
	bool *d_mask;
  	hipMalloc(&d_pose, 3 * P * sizeof(float)); 
  	hipMalloc(&d_max_pts, N * 3 * P * sizeof(float));
	hipMalloc(&d_hmap, H * W * sizeof(float));
	hipMalloc(&d_mask, H * W * sizeof(bool));

	hipMemcpy(d_pose, poses_inds, 3 * P * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_max_pts, max_pts_inds, N * 3 * P * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_hmap, hmap, H * W * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_mask, mask, H * W * sizeof(bool), hipMemcpyHostToDevice);

	int T = N * P;
	raytrace_k<<<GET_BLOCKS(T), CUDA_NUM_THREADS, 0, stream>>>(d_hmap, d_pose, d_max_pts, d_mask, N, W, H, P);

	// Read mask results
	hipMemcpy(mask, d_mask, H * W * sizeof(bool), hipMemcpyDeviceToHost);
	
	// error handling
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		std::cout << "CUDA kernel failed with error: " << hipGetErrorString(err);
	}
}
