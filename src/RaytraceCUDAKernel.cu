#include "hip/hip_runtime.h"
#include "gpu.cuh"
#include <iostream>

__global__ void raytrace_k(float *hmap, float *poses_inds, float *max_pts_inds,
                  		   float *scan, int N, int W, int H, int P, float res) {

	// Adapted from http://playtechs.blogspot.com/2007/03/raytracing-on-grid.html

	// Get indices
	int i = blockIdx.x * blockDim.x + threadIdx.x; // Ray index (one thread per ray) out of total rays
	int j = int(floor(i / N)); // Pose index
	int k = int(floor(i / P)); // Ray index for this specific scan
	if (i > (N * P)) return;

	/***** raytrace through ray (this is sloppy repetitive code but I'm lazy) *****/

	// Pull ending values for specific ray
	int x_max = (int)max_pts_inds[3*i];
	int y_max = (int)max_pts_inds[3*i+1];
	float z_max = max_pts_inds[3*i+2];

	// Pull pose values
	int pose_x = poses_inds[3*j];
	int pose_y = poses_inds[3*j+1];
	float pose_z = poses_inds[3*j+2];
	int z_start = int(floor(pose_z));
	
	// Setup
	int dx = abs(x_max - pose_x);
	int dy = abs(y_max - pose_y);
	float dz = fabs(z_max - pose_z);

	double dt_dx = 1.0 / dx;
	double dt_dy = 1.0 / dy;
	double dt_dz = 1.0 / dz;

	int n = 1;
	int x_inc, y_inc, z_inc;
	double t_next_x, t_next_y, t_next_z;

	// define initial variables based on cases
	// x
	if (dx == 0){
		x_inc = 0;
		t_next_x = 1000.0;
	} else if (x_max > pose_x) {
		x_inc = 1;
		n += x_max - pose_x;
		t_next_x = dt_dx;
	} else {
		x_inc = -1;
		n += pose_x - x_max;
		t_next_x = -dt_dx;
	}

	// y
	if (dy == 0){
		y_inc = 0;
		t_next_y = 1000.0;
	} else if (y_max > pose_y) {
		y_inc = 1;
		n += y_max - pose_y;
		t_next_y = dt_dy;
	} else {
		y_inc = -1;
		n += pose_y - y_max;
		t_next_y = -dt_dy;
	}

	// z
	if (dz == 0){
		z_inc = 0;
		t_next_z = 1000.0;
	} else if (z_max > z_start) {
		z_inc = 1;
		n += int(floor(z_max)) - z_start;
		t_next_z = (z_start + 1 - pose_z) * dt_dz;
	} else {
		z_inc = -1;
		n += z_start - int(floor(z_max));
		t_next_z = (pose_z - z_start) * dt_dz;
	}

	// loop through ray and update mask as necessary
	float z_curr = pose_z;
	int x_grid = pose_x;
	int y_grid = pose_y;
	int z_grid = z_start;
	float hmap_z, range;
	float x_out, y_out;
	double t = 0;
	float pose_x_m = res * pose_x;
	float pose_y_m = res * pose_y;

	for (; n > 0; --n){

		// check if current grid index is valid (return if not)
		if (x_grid >= W || x_grid < 0 || y_grid >= H || y_grid < 0) return;

		// Get current x, y, and z given t
		z_curr = pose_z + t * z_inc * dz;
		
		// check if current position is above ground (update scan and return if not)
		hmap_z = hmap[x_grid * W + y_grid];
		if (hmap_z >= z_curr && abs(t) > 0) {
			x_out = res * x_grid;
			y_out = res * y_grid;
			range = sqrt((x_out - pose_x_m) * (x_out - pose_x_m) + (y_out - pose_y_m) * (y_out - pose_y_m) + (hmap_z - pose_z) * (hmap_z - pose_z));
			scan[k * P + j] = range;
			// scan[j * N + k] = range;
			return;
		}

		// take a step along the ray
		if (t_next_x < t_next_y && t_next_x < t_next_z) {

			// x is min
			x_grid += x_inc;
			t = t_next_x;
			t_next_x += dt_dx;

		} else if(t_next_y < t_next_x && t_next_y < t_next_z) {

			// y is min
			y_grid += y_inc;
			t = t_next_y;
			t_next_y += dt_dy;

		} else {

			// z is min
			z_grid += z_inc;
			t = t_next_z;
			t_next_z += dt_dz;

		}
	}
}

void RaytraceCUDAKernel(float *hmap, float *poses_inds, float *max_pts_inds,
						float *scan, int N, int W, int H, int P, float res, hipStream_t stream) {
						
	// create shared arrays for heightmap and mask
	float *d_hmap, *d_pose, *d_max_pts, *d_scan;
  	int T = N * P;
	hipMalloc(&d_pose, 3 * P * sizeof(float)); 
  	hipMalloc(&d_max_pts, N * 3 * P * sizeof(float));
	hipMalloc(&d_hmap, H * W * sizeof(float));
	hipMalloc(&d_scan, T * sizeof(float));

	hipMemcpy(d_pose, poses_inds, 3 * P * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_max_pts, max_pts_inds, N * 3 * P * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_hmap, hmap, H * W * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_scan, scan, T * sizeof(float), hipMemcpyHostToDevice);

	raytrace_k<<<GET_BLOCKS(T), CUDA_NUM_THREADS, 0, stream>>>(d_hmap, d_pose, d_max_pts, d_scan, N, W, H, P, res);

	// Read mask results
	hipMemcpy(scan, d_scan, T * sizeof(float), hipMemcpyDeviceToHost);
	
	// error handling
	hipError_t err = hipGetLastError();
	if (hipSuccess != err){
		std::cout << "CUDA kernel failed with error: " << hipGetErrorString(err) << std::endl;
	}

	// Clear memory
	hipFree(d_pose);
	hipFree(d_max_pts);
	hipFree(d_hmap);
	hipFree(d_scan);

	d_pose=NULL;
	d_max_pts=NULL;
	d_hmap=NULL;
	d_scan=NULL;
}
